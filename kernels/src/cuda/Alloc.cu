#include <cuda/cuda_common.h>
#include <cuda/runtime.cuh>
#include <cuda/Alloc.h>

#ifdef __cplusplus
extern "C" {
#endif

void *alloc_cuda(const size_t size) {
    void *ptr;
    hipStream_t stream = get_cuda_stream();
    cudaCheck(hipMallocAsync((void**)&ptr, size, stream));
    return ptr;
}

void free_cuda(void *ptr) {
    if (ptr == NULL) return;
    hipStream_t stream = get_cuda_stream();
    cudaCheck(hipFreeAsync(ptr, stream));
}

#ifdef __cplusplus
}
#endif
