#include <cuda/cuda_common.h>
#include <cuda/runtime.h>
#include <cuda/runtime.cuh>
#include "utils.h"

hipblasHandle_t cublas_handle;
hipStream_t hip_stream;
int runtime_initialized = 0;
int runtime_destroyed = 0;
int runtime_cuda_stream_initialized = 0;
int runtime_cuda_stream_destroyed = 0;
int runtime_cublas_handle_initialized = 0;
int runtime_cublas_handle_destroyed = 0;

void runtime_init_cuda() {
    cudaCheck(hipStreamCreateWithFlags(&hip_stream, hipStreamNonBlocking));
    cublasCheck(hipblasCreate(&cublas_handle));
    // cublasCheck(hipblasSetMathMode(cublas_handle, HIPBLAS_TF32_TENSOR_OP_MATH));
    runtime_cuda_stream_initialized = 1;
    runtime_cublas_handle_initialized = 1;
    runtime_initialized = 1;
}

void runtime_destroy_cuda() {
    cublasCheck(hipblasDestroy(cublas_handle));
    cudaCheck(hipStreamDestroy(hip_stream));
    runtime_destroyed = 1;
    runtime_cuda_stream_destroyed = 1;
    runtime_cublas_handle_destroyed = 1;
}

hipStream_t get_cuda_stream() {
    CHECK_ERROR(runtime_cuda_stream_initialized == 0, "hip_stream has not been created. Please call runtime_cuda_init() for initializing the CUDA runtime environment.");
    CHECK_ERROR(runtime_cuda_stream_destroyed == 1, "hip_stream has been destoyed. Please call runtime_cuda_init() for initializing the CUDA runtime environment.");
    return hip_stream;
}

hipblasHandle_t get_cublas_handle() {
    CHECK_ERROR(runtime_cublas_handle_initialized == 0, "cublas_handle has not been created. Please call runtime_cuda_init() for initializing the CUDA runtime environment.");
    CHECK_ERROR(runtime_cublas_handle_destroyed == 1, "cublas_handle has been destoyed. Please call runtime_cuda_init() for initializing the CUDA runtime environment.");
    return cublas_handle;
}

void synchronize_cuda() {
    cudaCheck(hipDeviceSynchronize());
}