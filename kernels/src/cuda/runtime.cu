#include <cuda/cuda_common.h>
#include <cuda/runtime.h>

hipblasHandle_t cublas_handle;
int initialized = 0;

void setup_cublas_handle() {
    cublasCheck(hipblasCreate(&cublas_handle));
    cublasCheck(hipblasSetMathMode(cublas_handle, HIPBLAS_TF32_TENSOR_OP_MATH));
    initialized = 1;
}

hipblasHandle_t get_cublas_handle() {
    if (initialized == 0) setup_cublas_handle();
    return cublas_handle;
}