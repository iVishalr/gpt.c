#include "hip/hip_runtime.h"
#include <math.h>
#include <cuda/common.cuh>
#include <cuda/cuda_common.h>
#include <cuda/Alloc.h>
#include <cuda/Attention.h>
#include <cuda/Tensor.h>
#include <cuda/Softmax.h>
#include <cuda/runtime.h>
#include "utils.h"


C10_LAUNCH_BOUNDS_1(num_threads())
__global__ void apply_mask_cuda_kernel_impl(const float *mask, float *input, const int B, const int T, const int n_heads, const int ldmask) {
    const int tid = threadIdx.x;
    const int block_size = blockDim.x;
    const int i = blockIdx.x;

    float *att_tt = input + i * T * T;
    for (int j = tid; j < T; j += block_size) {
        float *att_j = att_tt + j * T;
        const float *mask_j = mask + j * ldmask;
        for (int k = 0; k < T; k++) {
            att_j[k] = mask_j[k] == 1.0f ? att_j[k] : -INFINITY;
        }
    }
}


C10_LAUNCH_BOUNDS_1(num_threads())
__global__ void permute_forward_cuda_kernel_impl(const float *input, float *q, float *k, float *v, const int B, const int T, const int n_heads, const int hs) {
    // okay so now, this kernel wants Q,K,V to all be of shape (B, n_heads, T, hs)
    // but instead, we have a single tensor QKV (input) of shape (B, T, 3, n_heads, hs)
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Q[b][nh_][n][d_] = input[b][n][0][nh_][d_]
    if (idx < B * n_heads * T * hs) {
        int b = idx / (n_heads * T * hs);
        int rest = idx % (n_heads * T * hs);
        int nh_ = rest / (T * hs);
        rest = rest % (T * hs);
        int n = rest / hs;
        int d_ = rest % hs;

        int inp_idx = \
            (b * T * 3 * n_heads * hs)
            +   (n * 3 * n_heads * hs)
            +       (0 * n_heads * hs)
            +          (nh_ * hs)
            +                d_;

        q[idx] = __ldcs(&input[inp_idx]);
        k[idx] = __ldcs(&input[inp_idx + n_heads * hs]);
        v[idx] = __ldcs(&input[inp_idx + 2 * (n_heads * hs)]);
    }
}


C10_LAUNCH_BOUNDS_1(num_threads())
__global__ void unpermute_forward_cuda_kernel_impl(const float *input, float *output, const int B, const int T, const int n_heads, const int hs) {
   // out has shape (B, nh, T, hs) but we need to unpermute it to (B, T, nh, hs)
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // out[b][n][nh_][d_] <- input[b][nh_][n][d_]
    if (idx < B * n_heads * T * hs) {
        int b = idx / (n_heads * T * hs);
        int rest = idx % (n_heads * T * hs);
        int nh_ = rest / (T * hs);
        rest = rest % (T * hs);
        int n = rest / hs;
        int d_ = rest % hs;

        int other_idx = (b * n_heads * T * hs) + (n * n_heads * hs) + (nh_ * hs) + d_;
        output[other_idx] = __ldcs(&input[idx]);
    }
}


C10_LAUNCH_BOUNDS_1(num_threads())
__global__ void permute_backward_cuda_kernel_impl(const float *dq, const float *dk, const float *dv, float *dout, const int B, const int T, const int n_heads, const int hs) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < B * n_heads * T * hs) {
        int b = idx / (n_heads * T * hs);
        int rest = idx % (n_heads * T * hs);
        int nh_ = rest / (T * hs);
        rest = rest % (T * hs);
        int n = rest / hs;
        int d_ = rest % hs;

        int inp_idx = (b * T * 3 * n_heads * hs) + (n * 3 * n_heads * hs) + (0 * n_heads * hs) + (nh_ * hs) + d_;
        dout[inp_idx] += dq[idx];
        dout[inp_idx + n_heads * hs] += dk[idx];
        dout[inp_idx + 2 * (n_heads * hs)] += dv[idx];
    }
}


C10_LAUNCH_BOUNDS_1(num_threads())
__global__ void unpermute_backward_cuda_kernel_impl(const float *global_grad, float *dout, int B, int T, int n_heads, int hs) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < B * n_heads * T * hs) {
        int b = idx / (n_heads * T * hs);
        int rest = idx % (n_heads * T * hs);
        int nh_ = rest / (T * hs);
        rest = rest % (T * hs);
        int n = rest / hs;
        int d_ = rest % hs;

        int other_idx = (b * n_heads * T * hs) + (n * n_heads * hs) + (nh_ * hs) + d_;
        dout[idx] += global_grad[other_idx];
    }
}


void apply_mask_cuda_kernel(const float *mask, float *input, const int B, const int T, const int n_heads, const int ldmask) {
    const int block_size = num_threads();
    const int grid_size = B * n_heads;
    apply_mask_cuda_kernel_impl<<<grid_size, block_size>>>(mask, input, B, T, n_heads, ldmask);
    cudaCheck(hipGetLastError());
}


void permute_forward_cuda_kernel(const float *input, float *q, float *k, float *v, const int B, const int T, const int C, const int n_heads) {
    const int hs = C / n_heads;
    const int block_size = num_threads();
    const int total_threads = B * T * n_heads * hs;
    const int grid_size = (total_threads + block_size - 1) / block_size;
    permute_forward_cuda_kernel_impl<<<grid_size, block_size>>>(input, q, k, v, B, T, n_heads, hs);
    cudaCheck(hipGetLastError());
}


void permute_backward_cuda_kernel(const float *dq, const float *dk, const float *dv, float *dout, const int B, const int T, const int C, const int n_heads) {
    const int hs = C / n_heads;
    const int block_size = num_threads();
    const int total_threads = B * T * n_heads * hs;
    const int grid_size = (total_threads + block_size - 1) / block_size;
    permute_backward_cuda_kernel_impl<<<grid_size, block_size>>>(dq, dk, dv, dout, B, T, n_heads, hs);
    cudaCheck(hipGetLastError());
}


void unpermute_forward_cuda_kernel(const float *input, float *output, const int B, const int T, const int C, const int n_heads) {
    const int block_size = num_threads();
    const int total_threads = B * T * C;
    const int grid_size = (total_threads + block_size - 1) / block_size;
    unpermute_forward_cuda_kernel_impl<<<grid_size, block_size>>>(input, output, B, T, n_heads, C / n_heads);
    cudaCheck(hipGetLastError());
}


void unpermute_backward_cuda_kernel(const float *global_grad, float *dout, const int B, const int T, const int C, const int n_heads) {
    const int hs = C / n_heads;
    const int block_size = num_threads();
    const int total_threads = B * T * C;
    const int grid_size = (total_threads + block_size - 1) / block_size;
    unpermute_backward_cuda_kernel_impl<<<grid_size, block_size>>>(global_grad, dout, B, T, n_heads, hs);
    cudaCheck(hipGetLastError());
}


#ifdef __cplusplus
extern "C" {
#endif

void attention_forward_cuda_kernel(
    const tensor_t *input,
    const tensor_t *mask,
    const int n_heads,
    tensor_t **cache,
    tensor_t *output
) {
    int B, T, C3, C, hs, mask_row_size;
    B = input->shape[0];
    T = input->shape[1];
    C3 = input->shape[2];
    C = C3 / 3;
    hs = C / n_heads;
    mask_row_size = mask->shape[mask->ndims - 1];

    const float scale = 1.0f / sqrtf(hs);
    tensor_t *k, *q, *v, *att;
    q = cache[0];
    k = cache[1];
    v = cache[2];
    att = cache[3];

    // permute 
    permute_forward_cuda_kernel(input->t, q->t, k->t, v->t, B, T, C, n_heads);
    
    // att = (q @ k.transpose(-2, -1)) * (1.0/sqrt(hs))
    sgemm_strided_batched_cuda(
        0, 1, 
        T, T, hs, 
        scale, 
        q, hs, T * hs, 
        k, hs, T * hs, 
        0.0f, 
        att, T, T * T, 
        B * n_heads
    );

    // apply mask
    apply_mask_cuda_kernel(mask->t, att->t, B, T, n_heads, mask_row_size);

    att->shape[1] = n_heads * T;
    softmax_forward_cuda_kernel(att, att);
    att->shape[1] = n_heads;

    // out = att @ v
    float *out = (float*)alloc_cuda(B * n_heads * T * hs * sizeof(float));
    float *tmp = output->t;
    output->t = out;
    sgemm_strided_batched_cuda(
        0, 0,
        T, hs, T,
        1.0f, 
        att, T, T * T,
        v, hs, T * hs,
        0.0f,
        output, hs, T * hs,
        B * n_heads
    );
    output->t = tmp;

    unpermute_forward_cuda_kernel(out, output->t, B, T, C, n_heads);
    free_cuda(out);
}


void attention_backward_cuda_kernel(
    const tensor_t *global_grad, 
    tensor_t **cache,
    const int n_heads,
    tensor_t *dout
) {
    int B, T, C, hs;
    B = global_grad->shape[0];
    T = global_grad->shape[1];
    C = global_grad->shape[2];
    hs = C / n_heads;

    const float scale = 1.0f / sqrtf(hs);
    const tensor_t *k, *q, *v, *att;
    q = cache[0];
    k = cache[1];
    v = cache[2];
    att = cache[3];

    tensor_t *dq, *dk, *dv, *datt, *dpreatt, *_global_grad;
    _global_grad = create_tensor(global_grad->shape, global_grad->ndims, CUDA);
    dq           = create_tensor(q->shape, q->ndims, CUDA);
    dk           = create_tensor(k->shape, k->ndims, CUDA);
    dv           = create_tensor(v->shape, v->ndims, CUDA);
    datt         = create_tensor(att->shape, att->ndims, CUDA);

    unpermute_backward_cuda_kernel(global_grad->t, _global_grad->t, B, T, C, n_heads);

    // datt = global_grad (B, n_heads, T, hs) @ v (B, n_heads, T, hs).T
    sgemm_strided_batched_cuda(
        0, 1, 
        T, T, hs,
        1.0f, 
        _global_grad, hs, T * hs,
        v, hs, T * hs,
        0.0f, 
        datt, T, T * T,
        B * n_heads
    );

    // dv = att (B, n_heads, T, T).T @ global_grad (B, n_heads, T, hs)
    sgemm_strided_batched_cuda(
        1, 0, 
        T, hs, T,
        1.0f, 
        att, T, T * T,
        _global_grad, hs, T * hs,
        0.0f, 
        dv, hs, T * hs,
        B * n_heads
    );

    datt->shape[1] = n_heads * T;
    softmax_backward_cuda_kernel(datt, att, datt);
    datt->shape[1] = n_heads;

    // dq = dpreatt (B, n_heads, T, T) @ k (B, n_heads, T, hs)
    sgemm_strided_batched_cuda(
        0, 0, 
        T, hs, T,
        scale, 
        datt, T, T * T,
        k, hs, T * hs,
        0.0f, 
        dq, hs, T * hs,
        B * n_heads
    );

    // dk = dpreatt (B, n_heads, T, T) @ q (B, n_heads, T, hs)
    sgemm_strided_batched_cuda(
        1, 0, 
        T, hs, T,
        scale, 
        datt, T, T * T,
        q, hs, T * hs,
        0.0f, 
        dk, hs, T * hs,
        B * n_heads
    );

    permute_backward_cuda_kernel(dq->t, dk->t, dv->t, dout->t, B, T, C, n_heads);

    free_tensor(dq);
    free_tensor(dk);
    free_tensor(dv);
    free_tensor(datt);
    free_tensor(_global_grad);
}

#ifdef __cplusplus
}
#endif
