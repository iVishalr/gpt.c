#include <cpu/Alloc.h>
#include <cuda/Alloc.h>
#include <cuda/Tensor.h>
#include <cuda/cuda_common.h>
#include <cuda/runtime.h>

#include <common/kutils.h>

#ifdef __cplusplus
extern "C" {
#endif

void move_tensor_to_host_cuda(tensor_t *tensor) {
    CHECK_ERROR(tensor == NULL, "Expected *tensor to be a tensor_t pointer. Got NULL");
    CHECK_ERROR(tensor->t == NULL, "Expected *tensor->t to be a float pointer. Got NULL");

    // if tensor is already present on host, return
    if (tensor->device == CPU) return;

    float *device_ptr = tensor->t;
    float *host_ptr = (float*)aligned_alloc_cpu(tensor->length * sizeof(float), 64);
    cudaCheck(hipMemcpy(host_ptr, device_ptr, tensor->length * sizeof(float), hipMemcpyDeviceToHost));

    tensor->t = host_ptr;
    tensor->device = CPU;
    free_cuda(device_ptr);
}

void move_tensor_to_device_cuda(tensor_t *tensor) {
    CHECK_ERROR(tensor == NULL, "Expected *tensor to be a tensor_t pointer. Got NULL");
    CHECK_ERROR(tensor->t == NULL, "Expected *tensor->t to be a float pointer. Got NULL");

    // if tensor is already present on device, return
    if (tensor->device == 1) {
        printf("Tensor already on CUDA. Returning.\n");
        return;
    }

    float *host_ptr = tensor->t;
    float *device_ptr;
    const size_t size = tensor->length * sizeof(float);
    cudaCheck(hipMalloc((void**)&device_ptr, size));
    cudaCheck(hipMemcpy(device_ptr, host_ptr, size, hipMemcpyHostToDevice));

    tensor->t = device_ptr;
    tensor->device = CUDA;
    free_cpu(host_ptr);
}

void create_tensor_data_cuda(tensor_t *tensor) {
    CHECK_ERROR(tensor == NULL, "Expected *tensor to be a tensor_t pointer. Got NULL");
    CHECK_ERROR(tensor->t != NULL, "Expected *tensor->t to be NULL. Got a pointer to address %p.", tensor->t);
    tensor->t = (float*)alloc_cuda(tensor->length * sizeof(float));
}

void zeros_tensor_data_cuda(tensor_t *tensor) {
    CHECK_ERROR(tensor == NULL, "Expected *tensor to be a tensor_t pointer. Got NULL");
    if (!tensor->t) create_tensor_data_cuda(tensor);
    cudaCheck(hipMemset(tensor->t, 0, tensor->length * sizeof(float)));
}

void sgemm_cuda(
    const int TransA, const int TransB, const int M, const int N, const int K,
    const float alpha, const tensor_t *A, const int offsetA, const int lda,
    const tensor_t *B, const int offsetB, const int ldb, 
    const float beta, tensor_t *C, const int offsetC, const int ldc
) {

    float *_A = A->t;
    float *_B = B->t;
    float *_C = C->t;

    hipblasOperation_t transa = (TransA == 1) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    hipblasOperation_t transb = (TransB == 1) ? HIPBLAS_OP_T : HIPBLAS_OP_N;

    // Adjust pointers with offsets if provided
    const float *A_ptr = _A + offsetA;
    const float *B_ptr = _B + offsetB;
    float *C_ptr = _C + offsetC;

    hipblasHandle_t cublas_handle = get_cublas_handle();

    // cuBLAS gemm: C = alpha * op(A) * op(B) + beta * C
    cublasCheck(
        hipblasSgemm(
            cublas_handle, transb, transa,  // Note swapped order for row-major
            N, M, K,                        // Dimensions
            &alpha,                         // Scalar alpha
            B_ptr, ldb,                     // Matrix B
            A_ptr, lda,                     // Matrix A
            &beta,                          // Scalar beta
            C_ptr, ldc                      // Matrix C
        )
    );
}

void sgemm_strided_batched_cuda(
    const int TransA, const int TransB, const int M, const int N, const int K,
    const float alpha, const tensor_t *A, const int lda, const int strideA,
    const tensor_t *B, const int ldb, const int strideB,
    const float beta, tensor_t *C, const int ldc, const int strideC, const int batch_count
) {
    const float *_A = A->t;
    const float *_B = B->t;
    float *_C = C->t;

    hipblasOperation_t transa = (TransA == 1) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    hipblasOperation_t transb = (TransB == 1) ? HIPBLAS_OP_T : HIPBLAS_OP_N;

    hipblasHandle_t cublas_handle = get_cublas_handle();

    // cuBLAS gemm: C = alpha * op(A) * op(B) + beta * C
    cublasCheck(
        hipblasSgemmStridedBatched(
            cublas_handle, transb, transa, // Note swapped order for row-major
            N, M, K,                       // Dimensions
            &alpha,                        // Scalar alpha
            _B, ldb, strideB,              // Matrix B
            _A, lda, strideA,              // Matrix A
            &beta,                         // Scalar beta
            _C, ldc, strideC,              // Matrix C
            batch_count                    // batch_count
        )
    );
}

#ifdef __cplusplus
}
#endif