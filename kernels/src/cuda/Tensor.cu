#include <cpu/Alloc.h>
#include <cuda/Alloc.h>
#include <cuda/Tensor.h>
#include <cuda/cuda_common.h>
#include <cuda/runtime.h>

#include <common/kutils.h>

#ifdef __cplusplus
extern "C" {
#endif

void move_tensor_to_host_cuda(tensor_t *tensor) {
    CHECK_ERROR(tensor == NULL, "Expected *tensor to be a tensor_t pointer. Got NULL");
    CHECK_ERROR(tensor->t == NULL, "Expected *tensor->t to be a float pointer. Got NULL");

    // if tensor is already present on host, return
    if (tensor->device == CPU) return;

    float *device_ptr = tensor->t;
    float *host_ptr = (float*)AllocCheck(aligned_alloc_cpu, tensor->length * sizeof(float), 64);
    cudaCheck(hipMemcpy(host_ptr, device_ptr, tensor->length * sizeof(float), hipMemcpyDeviceToHost));

    tensor->t = host_ptr;
    tensor->device = CPU;
    free_cuda(device_ptr);
}

void move_tensor_to_device_cuda(tensor_t *tensor) {
    CHECK_ERROR(tensor == NULL, "Expected *tensor to be a tensor_t pointer. Got NULL");
    CHECK_ERROR(tensor->t == NULL, "Expected *tensor->t to be a float pointer. Got NULL");

    // if tensor is already present on device, return
    if (tensor->device == CUDA) return;

    float *host_ptr = tensor->t;
    float *device_ptr;
    const size_t size = tensor->length * sizeof(float);
    cudaCheck(hipMalloc((void**)&device_ptr, size));
    cudaCheck(hipMemcpy(device_ptr, host_ptr, size, hipMemcpyHostToDevice));

    tensor->t = device_ptr;
    tensor->device = CUDA;
    free_cpu(host_ptr);
}

void sgemm_cuda(
    const int TransA, const int TransB, const int M, const int N, const int K,
    const float alpha, const tensor_t *A, const int offsetA, const int lda,
    const tensor_t *B, const int offsetB, const int ldb, 
    const float beta, tensor_t *C, const int offsetC, const int ldc
) {

    float *_A = A->t;
    float *_B = B->t;
    float *_C = C->t;

    hipblasOperation_t transa = (TransA == 1) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    hipblasOperation_t transb = (TransB == 1) ? HIPBLAS_OP_T : HIPBLAS_OP_N;

    // Adjust pointers with offsets if provided
    const float *A_ptr = _A + offsetA;
    const float *B_ptr = _B + offsetB;
    float *C_ptr = _C + offsetC;

    hipblasHandle_t cublas_handle = get_cublas_handle();

    // // cuBLAS gemm: C = alpha * op(A) * op(B) + beta * C
    cublasCheck(
        hipblasSgemm(
            cublas_handle, transb, transa,  // Note swapped order for row-major
            N, M, K,                        // Dimensions
            &alpha,                         // Scalar alpha
            B_ptr, ldb,                     // Matrix B
            A_ptr, lda,                     // Matrix A
            &beta,                          // Scalar beta
            C_ptr, ldc                      // Matrix C
        )
    );
}

#ifdef __cplusplus
}
#endif